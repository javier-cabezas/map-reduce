#include "hip/hip_runtime.h"
/*
 * <+ DESCRIPTION +>
 *
 * Copyright (C) 2011, Javier Cabezas <jcabezas in ac upc edu> {{{
 *
 * This program is free software; you can redistribute it and/or 
 * modify it under the terms of the GNU General Public License 
 * as published by the Free Software Foundation; either 
 * version 2 of the License, or any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 675 Mass Ave, Cambridge, MA 02139, USA.
 * }}}
 */

#if 0

#include <cstdio>

#include "gmap"

const unsigned a = 1024;
const unsigned b = 1024;

static float array1[a][b];
static float array2[a][b];

static float array3[a][b];

#define USE_LAMBDA

int main(int argc, char *argv[])
{
#ifdef USE_LAMBDA
    independent(a, b,
        [&array1, &array2](unsigned y, unsigned x)
        {
#else
    #pragma omp parallel for
    for (unsigned y = 0; y < a; y++) {
        for (unsigned x = 0; x < b; x++) {
#endif
            array1[y][x] = y * 10 + x;
            array2[y][x] = y * 20 + x;

#ifdef USE_LAMBDA
        });
#else
        }
    }
#endif

#ifdef USE_LAMBDA
    independent(a, b,
        [&array3, &array1, &array2](unsigned y, unsigned x)
        {
#else
    #pragma omp parallel for
    for (unsigned y = 0; y < a; y++) {
        for (unsigned x = 0; x < b; x++) {
#endif
            float val = 0.f;
            for (unsigned i = 0; i < a; i++) {
                 val += array1[y][i] * array2[i][x];
            }
            array3[y][x] = val;
#ifdef USE_LAMBDA
        });
#else
        }
    }
#endif

    float accum = 0.f;
#ifdef USE_LAMBDA
    sequential(a, b,
        [&accum, &array3](unsigned y, unsigned x)
        {
#else
    for (unsigned y = 0; y < a; y++) {
        for (unsigned x = 0; x < b; x++) {
#endif
            accum += array3[y][x];
#ifdef USE_LAMBDA
        });
#else
        }
    }
#endif

    printf("Total: %f\n", accum);

    return 0;
}
#endif

#include <cassert>
#include <cstdio>

#define C1 2.3f
#define C2 3.2f
#define C3 1.7f
#define C4 4.9f

template <typename Func>
__global__
void kernel(Func f, float *a)
{
    f(threadIdx.x, threadIdx.y, a);
}

float a_host, *a_dev;

template <typename Func>
void map(Func f, int x, int y)
{
    if (1) {
        kernel<<<1, 1>>>(f, a_dev);
        hipError_t err = hipDeviceSynchronize();
        assert(err == hipSuccess);
    }
}

typedef void (*func)(int x, int y, float *a);

__device__
void func1(int x, int y, float *a)
{
    *a = 69.f;
}

__device__
func func1_p = func1;

int main(int argc, char *argv[])
{
    hipError_t err = hipMalloc(&a_dev, sizeof(float));
    assert(err == hipSuccess);

    func f;
    hipMemcpyFromSymbol(&f, HIP_SYMBOL(func1_p), sizeof(func));

    map(f, 1000, 1000);

    err = hipMemcpy(&a_host, a_dev, sizeof(float), hipMemcpyDefault);
    assert(err == hipSuccess);

    printf("%f\n", a_host);

    return 0;
}

/* vim:set backspace=2 tabstop=4 shiftwidth=4 textwidth=120 foldmethod=marker expandtab: */
